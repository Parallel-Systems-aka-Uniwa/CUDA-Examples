#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
 

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int k, id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    for (k = id; k < n; k += blockDim.x*gridDim.x)
      {
        c[k] = a[k] + b[k];
      }
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 50000;
 
    // Size of vectors
    int blockSize = 1024;
    int gridSize = 10;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    hipEvent_t start,stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    // Number of threads in each thread block
    //blockSize = 1024;
 
    // Number of thread blocks in grid
    //gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    printf("CUDA kernel launch with %d blocks of %d threads\n", gridSize, blockSize);
    
    hipEventRecord(start,0);
    
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
 
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);
 
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf ("Time for the kernel: %f ms\n", elapsedTime);
    
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
