
#include <hip/hip_runtime.h>
__global__ void add(int *a) {
	*a = *a + 1;
}

int main(void) {
	int a; 		// αντίγραφο του a για τον host
	int *d_a; 	// αντίγραφο του a για την device
	int size = sizeof(int);
	// Εκχώρηση μνήμης για το αντίγραφο a στην συσκευή
	hipMalloc((void **)&d_a, size); 
	// Ορισμός αρχικής τιμής
	a = 2;
	// Αντιγραφή από host σε device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	// Κλήση add() (συνάρτηση πυρήνα) στην GPU
	add<<<1,1>>>(d_a);
	// Αντιγραφή από device σε host
	hipMemcpy(&a, d_a, size, hipMemcpyDeviceToHost);
	// Εκκαθάριση / απελευθέρωση μνήμης
	hipFree(d_a);
	return 0;
}
