
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int *a_d) {
	*a_d += 1;
}

int main() {
        int a=0, *a_d;
        hipMalloc((void**) &a_d, sizeof(int));
        hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);
        kernel<<<1000,1000>>>(a_d);
        hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);
        printf("a = %d\n", a);
        hipFree(a_d);
}
