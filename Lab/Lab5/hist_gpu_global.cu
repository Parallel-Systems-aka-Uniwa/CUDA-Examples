
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE    (100*1024*1024)

void* big_random_block( int size ) {
    unsigned char *data = (unsigned char*)malloc( size );
    for (int i=0; i<size; i++)
        data[i] = rand();
    return data;
}

__global__ void histo_kernel( unsigned char *buffer,
                              long size,
                              unsigned int *histo ) {
    // calculate the starting idx and the offset to the next
    // block that each thread will be processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd( &histo[buffer[i]], 1 );
        i += stride;
    }
}

int main( void ) {
    unsigned char *buffer =
                    (unsigned char*)big_random_block( SIZE );

    hipEvent_t     start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // allocate memory on the GPU for the file's data
    unsigned char *dev_buffer; unsigned int *dev_histo;
    hipMalloc((void**)&dev_buffer, SIZE);
    hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);
    hipMalloc((void**)&dev_histo, 256 * sizeof(int));
    hipMemset(dev_histo, 0, 256 * sizeof(int));

    // kernel launch - 2x the number of mps gave best timing
    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0);
    int blocks = prop.multiProcessorCount;
    histo_kernel<<<blocks*2,256>>>( dev_buffer, SIZE, dev_histo );
    unsigned int    histo[256];
    hipMemcpy(histo, dev_histo, 256*sizeof(int),hipMemcpyDeviceToHost);

    // get stop time, and display the timing results
    hipEventRecord(stop, 0); hipEventSynchronize(stop);
    float   elapsedTime; hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to generate:  %3.1f ms\n", elapsedTime);

    long histoCount = 0;
    for (int i=0; i<256; i++) { histoCount += histo[i]; }
    printf( "Histogram Sum:  %ld\n", histoCount );
    // verify that we have the same counts via CPU
    for (int i=0; i<SIZE; i++) histo[buffer[i]]--;
    for (int i=0; i<256; i++) {
        if (histo[i] != 0)
            printf( "Failure at %d!  Off by %d\n", i, histo[i] );
    }
    hipEventDestroy(start);  hipEventDestroy(stop);
    hipFree(dev_histo); hipFree(dev_buffer);
    free(buffer); return 0;
}